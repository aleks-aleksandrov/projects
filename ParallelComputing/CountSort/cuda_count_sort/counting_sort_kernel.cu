#include "hip/hip_runtime.h"
/* Write GPU code to perform the step(s) involved in counting sort. 
 Add additional kernels and device functions as needed. */

__global__ void histogram_generation_kernel(int *input_data, int *histogram, 
                                int num_elements, int histogram_size)
{
    /*STEP#1: Generate the histogram from the input data in shared memory*/

    __shared__ unsigned int shared_histogram[HISTOGRAM_SIZE];

    /* Initialize shared memory */ 
    if(threadIdx.x < histogram_size)
        shared_histogram[threadIdx.x] = 0;
    
    __syncthreads();

    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    
    while (offset < num_elements) {
        atomicAdd(&shared_histogram[input_data[offset]], 1);
        offset += stride;
    }
    __syncthreads();

    /*Transfer histogram to global memory*/
    if (threadIdx.x < histogram_size) 
        atomicAdd(&histogram[threadIdx.x], shared_histogram[threadIdx.x]);

    return;
}


//n - histogram size, scan_out is used for testin - remove on final submission
__global__ void counting_sort_kernel(int *sorted_array, int *histogram, int n)
{   
    //STEP #2 - perform a scan
    __shared__ int scan_shared[2*HISTOGRAM_SIZE];

    int tid = threadIdx.x;
    int offset;

    /* Indices for the ping-pong buffers */
    int pout = 0;
    int pin = 1;


    //inclusive scan
    scan_shared[pout * n + tid] = histogram[tid];

    /*Perform Inclusive Scan*/
    for (offset = 1; offset < n; offset *= 2) {
        pout = 1 - pout;
        pin  = 1 - pout;
        __syncthreads();

        //coppies array to the second half of scanned array
        scan_shared[pout * n + tid] = scan_shared[pin * n + tid]; 

        if (tid >= offset)
            scan_shared[pout * n + tid] += scan_shared[pin * n + tid - offset];
    }

    __syncthreads();

    
    //STEP#3 - generate the sorted array
    int j;
    int start_idx = 0;

    if(tid > 0)
        start_idx = scan_shared[tid-1];

    // printf("Histogram = %d, Scanned IDX = %d, Bin = %d \n",  histogram[tid],start_idx,tid);
    for (j = start_idx; j < (start_idx+histogram[tid]); j++)
        sorted_array[j] = tid;
    
    
    return;
}