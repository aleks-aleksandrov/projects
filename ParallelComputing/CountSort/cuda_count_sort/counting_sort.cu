#include "hip/hip_runtime.h"
/* Host-side code to perform counting sort 
 * 
 * Author: Naga Kandasamy
 * Date modified: March 8, 2022
 * 
 * Student name(s): FIXME
 * Date modified: FIXME
 * 
 * Compile as follows: make clean && make
 */

#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <string.h>
#include <math.h>
#include <limits.h>


/* Do not change the range value */
#define MIN_VALUE 0 
#define MAX_VALUE 255

#define THREAD_BLOCK_SIZE 256 
#define HISTOGRAM_SIZE 256 

#include "counting_sort_kernel.cu"

// #define THREAD_BLOCK_SIZE 11 
// #define HISTOGRAM_SIZE 11 

/* Uncomment to spit out debug info */
//#define DEBUG

extern "C" int counting_sort_gold(int *, int *, int, int);
int rand_int(int, int);
void print_array(int *, int);
void print_min_and_max_in_array(int *, int);
void compute_on_device(int *, int *, int, int);
void compute_on_device_not_optimized(int *, int *, int, int);
void compute_on_device_optimized(int *, int *, int, int);
int check_if_sorted(int *, int);
int compare_results(int *, int *, int);

int main(int argc, char **argv)
{
    if (argc < 2) {
        printf("Usage: %s num-elements\n", argv[0]);
        exit(EXIT_FAILURE);
    }

    int num_elements = atoi(argv[1]);
    int range = MAX_VALUE - MIN_VALUE;
    int *input_array, *sorted_array_reference, *sorted_array_d;

    struct timeval start, stop;	

    /* Populate input array with random integers between [0, RANGE] */
    printf("Generating input array with %d elements in the range 0 to %d\n", num_elements, range);
    input_array = (int *)malloc(num_elements * sizeof(int));
    if (input_array == NULL) {
        perror("malloc");
        exit(EXIT_FAILURE);
    }
    
    srand(time(NULL));
    int i;
    for (i = 0; i < num_elements; i++)
        input_array[i] = rand_int (MIN_VALUE, MAX_VALUE);

#ifdef DEBUG
    print_array(input_array, num_elements);
    print_min_and_max_in_array(input_array, num_elements);
#endif

    /* Sort elements in input array using reference implementation. 
     * The result is placed in sorted_array_reference. */
    printf("\nSorting array on CPU\n");
    int status;
    sorted_array_reference = (int *)malloc(num_elements * sizeof(int));
    if (sorted_array_reference == NULL) {
        perror("malloc"); 
        exit(EXIT_FAILURE);
    }
    memset(sorted_array_reference, 0, num_elements);

    gettimeofday(&start, NULL);
    status = counting_sort_gold(input_array, sorted_array_reference, num_elements, range);
    gettimeofday(&stop, NULL);
	fprintf(stderr, "Execution time CPU= %fs\n", (float)(stop.tv_sec - start.tv_sec +\
                (stop.tv_usec - start.tv_usec)/(float)1000000));

    if (status == -1) {
        exit(EXIT_FAILURE);
    }

    status = check_if_sorted(sorted_array_reference, num_elements);
    if (status == -1) {
        printf("Error sorting the input array using the reference code\n");
        exit(EXIT_FAILURE);
    }

    printf("Counting sort was successful on the CPU\n");

#ifdef DEBUG
    print_array(sorted_array_reference, num_elements);
#endif

    /* FIXME: Write function to sort elements in the array in parallel fashion. 
     * The result should be placed in sorted_array_mt. */
    printf("\nSorting array on GPU\n");
    sorted_array_d = (int *)malloc(num_elements * sizeof(int));
    if (sorted_array_d == NULL) {
        perror("malloc");
        exit(EXIT_FAILURE);
    }
    memset(sorted_array_d, 0, num_elements);

    // test_input[10] = {8,5,1,3,7,8,6,5,3,8};
    // compute_on_device(test_input, sorted_array_d, 10, range);

    gettimeofday(&start, NULL);
    compute_on_device(input_array, sorted_array_d, num_elements, range);
    //compute_on_device(test_input, sorted_array_d, 10, range);
    //compute_on_device_not_optimized(input_array, sorted_array_d, num_elements, range);
    gettimeofday(&stop, NULL);
	fprintf(stderr, "Execution time GPU= %fs\n", (float)(stop.tv_sec - start.tv_sec +\
                (stop.tv_usec - start.tv_usec)/(float)1000000));

    /* Check the two results for correctness */
    printf("\nComparing CPU and GPU results\n");
    status = compare_results(sorted_array_reference, sorted_array_d, num_elements);
    if (status == 0)
        printf("Test passed\n");
    else
        printf("Test failed\n");

    exit(EXIT_SUCCESS);
}

/* FIXME: Write the GPU implementation of counting sort */
void compute_on_device(int *input_array, int *sorted_array, int num_elements, int range)
{
    
    /* Allocate space on GPU for input data */
    int *input_array_on_device = NULL;
    hipMalloc((void**)&input_array_on_device, num_elements * sizeof(int));
	hipMemcpy(input_array_on_device, input_array, num_elements * sizeof(int), hipMemcpyHostToDevice);

    /* Allocate space on GPU for output data */
    int *sorted_array_device = NULL;
    hipMalloc((void**)&sorted_array_device, num_elements * sizeof(int));

    /* Allocate space on GPU for histogram data */
    int *histogram_on_device = NULL;
    hipMalloc((void**)&histogram_on_device, HISTOGRAM_SIZE * sizeof(int));
	hipMemset(histogram_on_device, 0, HISTOGRAM_SIZE * sizeof(int));

    /* Set up the execution grid on GPU */
	dim3 thread_block(THREAD_BLOCK_SIZE, 1, 1);
    printf("Generating grid dimension of %d blocks, %d threads", num_elements/THREAD_BLOCK_SIZE, THREAD_BLOCK_SIZE);
	dim3 grid(num_elements/THREAD_BLOCK_SIZE,1);

    //generate histogram
    histogram_generation_kernel<<<grid, thread_block>>>(input_array_on_device, histogram_on_device, num_elements, HISTOGRAM_SIZE); 
    hipDeviceSynchronize();

    dim3 grid_2(1,1);
    //Scan and sort array
    counting_sort_kernel<<< grid_2, thread_block>>>(sorted_array_device,histogram_on_device, HISTOGRAM_SIZE);
    hipDeviceSynchronize();

    //Copy result to CPU
    hipMemcpy(sorted_array, sorted_array_device, num_elements * sizeof(int), hipMemcpyDeviceToHost);

    // /* Free memory */
	hipFree(input_array_on_device);
	hipFree(histogram_on_device);
    hipFree(sorted_array_device);
    return;
}

/* FIXME: Write optimized GPU implementation of counting sort */
void compute_on_device_optimized(int *input_array, int *sorted_array, int num_elements, int range)
{
     return;
}


/* Check if array is sorted */
int check_if_sorted(int *array, int num_elements)
{
    int status = 0;
    int i;
    for (i = 1; i < num_elements; i++) {
        if (array[i - 1] > array[i]) {
            status = -1;
            break;
        }
    }

    return status;
}

/* Check if the arrays elements are identical */ 
int compare_results(int *array_1, int *array_2, int num_elements)
{
    int status = 0;
    int i;
    for (i = 0; i < num_elements; i++) {
        if (array_1[i] != array_2[i]) {
            status = -1;
            break;
        }
    }

    return status;
}

/* Return random integer between [min, max] */ 
int rand_int(int min, int max)
{
    float r = rand()/(float)RAND_MAX;
    return (int)floorf(min + (max - min) * r);
}

/* Print given array */
void print_array(int *this_array, int num_elements)
{
    printf("Array: ");
    int i;
    for (i = 0; i < num_elements; i++)
        printf("%d ", this_array[i]);
    
    printf("\n");
    return;
}

/* Return min and max values in given array */
void print_min_and_max_in_array(int *this_array, int num_elements)
{
    int i;

    int current_min = INT_MAX;
    for (i = 0; i < num_elements; i++)
        if (this_array[i] < current_min)
            current_min = this_array[i];

    int current_max = INT_MIN;
    for (i = 0; i < num_elements; i++)
        if (this_array[i] > current_max)
            current_max = this_array[i];

    printf("Minimum value in the array = %d\n", current_min);
    printf("Maximum value in the array = %d\n", current_max);
    return;
}


