#include "hip/hip_runtime.h"
/* Reference code implementing the box blur filter.

    Build and execute as follows: 
        make clean && make 
        ./blur_filter size

    Author: Naga Kandasamy
    Date modified: February 16, 2022

    Student name(s): FIXME
    Date modified: FIXME
*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>

// #define DEBUG

/* Include the kernel code */
#include "blur_filter_kernel.cu"

extern "C" void compute_gold(const image_t, image_t);
void compute_on_device(const image_t, image_t);
int check_results(const float *, const float *, int, float);
void print_image(const image_t);

//Compute on Device Functions
image_s allocate_matrix_on_device(image_s I);
void copy_matrix_to_device(image_s Idevice, image_s Ihost);
void copy_matrix_from_device(image_s Ihost, image_s Idevice);

struct timeval start, stop;	

int main(int argc, char **argv)
{
    if (argc < 2) {
        fprintf(stderr, "Usage: %s size\n", argv[0]);
        fprintf(stderr, "size: Height of the image. The program assumes size x size image.\n");
        exit(EXIT_FAILURE);
    }

    /* Allocate memory for the input and output images */
    int size = atoi(argv[1]);

    fprintf(stderr, "Creating %d x %d images\n", size, size);
    image_t in, out_gold, out_gpu;
    in.size = out_gold.size = out_gpu.size = size;
    in.element = (float *)malloc(sizeof(float) * size * size);
    out_gold.element = (float *)malloc(sizeof(float) * size * size);
    out_gpu.element = (float *)malloc(sizeof(float) * size * size);
    if ((in.element == NULL) || (out_gold.element == NULL) || (out_gpu.element == NULL)) {
        perror("Malloc");
        exit(EXIT_FAILURE);
    }

    /* Poplulate our image with random values between [-0.5 +0.5] */
    srand(time(NULL));
    int i;
    for (i = 0; i < size * size; i++)
        in.element[i] = rand()/(float)RAND_MAX -  0.5;
  
   /* Calculate the blur on the CPU. The result is stored in out_gold. */
    fprintf(stderr, "Calculating blur on the CPU\n"); 

    gettimeofday(&start, NULL);
    compute_gold(in, out_gold); 
    gettimeofday(&stop, NULL);
    fprintf(stderr, "Execution time = %fs\n", (float)(stop.tv_sec - start.tv_sec +\
        (stop.tv_usec - start.tv_usec)/(float)1000000));

#ifdef DEBUG 
   print_image(in);
   print_image(out_gold);
#endif

    /* FIXME: Calculate the blur on the GPU. The result is stored in out_gpu. */
    fprintf(stderr, "Calculating blur on the GPU\n");

    //gettimeofday(&start, NULL);
    compute_on_device(in, out_gpu);
    //gettimeofday(&stop, NULL);
    //fprintf(stderr, "Execution time = %fs\n", (float)(stop.tv_sec - start.tv_sec + (stop.tv_usec - start.tv_usec)/(float)1000000));

    /* Check CPU and GPU results for correctness */
    fprintf(stderr, "Checking CPU and GPU results\n");
    int num_elements = out_gold.size * out_gold.size;
    float eps = 1e-6;    /* Do not change */
    int check;

        //print_image(out_gold);
        //print_image(out_gpu);
    check = check_results(out_gold.element, out_gpu.element, num_elements, eps);
    if (check == 0) 
        fprintf(stderr, "TEST PASSED\n");
    else
        fprintf(stderr, "TEST FAILED\n");
    
    /* Free data structures on the host */
    free((void *)in.element);
    free((void *)out_gold.element);
    free((void *)out_gpu.element);

    exit(EXIT_SUCCESS);
}

/* FIXME: Complete this function to calculate the blur on the GPU */
void compute_on_device(const image_t in, image_t out)
{   

    /* Allocate memory and copy matrices to device */ 
    image_t I_d = allocate_matrix_on_device(in);

    //Copy values to device
    copy_matrix_to_device(I_d,in);

    //Allocate memory for the output image matrix
    image_t I_Out_d = allocate_matrix_on_device(out);

    /* Set up the execution grid */
    dim3 threads(TILE_SIZE, TILE_SIZE);

    fprintf(stderr, "Setting up a %d x %d grid of thread blocks\n", 
            (in.size + TILE_SIZE - 1)/TILE_SIZE, (in.size + TILE_SIZE - 1)/TILE_SIZE);
    dim3 grid((in.size)/TILE_SIZE, (in.size)/TILE_SIZE);

    /* Launch kernel */
    gettimeofday(&start, NULL);
    blur_filter_kernel<<< grid, threads >>>(I_d.element, I_Out_d.element, in.size);
    gettimeofday(&stop, NULL);
    
    fprintf(stderr, "Execution time = %fs\n", (float)(stop.tv_sec - start.tv_sec + (stop.tv_usec - start.tv_usec)/(float)1000000));
    
    hipDeviceSynchronize();
    
    //Copy the matrix back from the GPU
    copy_matrix_from_device(out, I_Out_d);     

    return;
}

/* Check correctness of results */
int check_results(const float *pix1, const float *pix2, int num_elements, float eps) 
{
    int i;
    for (i = 0; i < num_elements; i++)
        if (fabsf((pix1[i] - pix2[i])/pix1[i]) > eps) 
            return -1;
    
    return 0;
}

/* Print out the image contents */
void print_image(const image_t img)
{
    int i, j;
    float val;
    for (i = 0; i < img.size; i++) {
        for (j = 0; j < img.size; j++) {
            val = img.element[i * img.size + j];
            printf("%0.4f ", val);
        }
        printf("\n");
    }

    printf("\n");
}


/* Allocate memory on device for matrix */
image_s allocate_matrix_on_device(image_s I)                        
{
	image_s Idevice = I;
	int size = I.size * I.size * sizeof(float);
	
    hipMalloc((void**)&Idevice.element, size);
    if (Idevice.element == NULL) {
        fprintf(stderr, "CudaMalloc error\n");
        exit(EXIT_FAILURE);
    }

	return Idevice;
}

/* Copy matrix from host memory to device memory */
void copy_matrix_to_device(image_s Idevice, image_s Ihost)      
{
	int size = Ihost.size * Ihost.size * sizeof(float);
	hipMemcpy(Idevice.element, Ihost.element, size, hipMemcpyHostToDevice);
}

/* Copy matrix from device memory to host memory */
void copy_matrix_from_device (image_s Ihost, image_s Idevice)   
{   
	int size = Idevice.size * Idevice.size * sizeof(float);
	hipMemcpy(Ihost.element, Idevice.element, size, hipMemcpyDeviceToHost);
}
